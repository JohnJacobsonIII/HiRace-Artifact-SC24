#include "hip/hip_runtime.h"
// BUG IN SRAD APPLICATIONS SEEMS TO BE SOMEWHERE IN THIS CODE, MEMORY CORRUPTION

// srad kernel
__global__ void srad2(	fp d_lambda, 
										int d_Nr, 
										int d_Nc, 
										long d_Ne, 
										int *_d_iN, 
										int *_d_iS, 
										int *_d_jE, 
										int *_d_jW,
										fp *_d_dN, 
										fp *_d_dS, 
										fp *_d_dE, 
										fp *_d_dW, 
										fp *_d_c, 
										fp *_d_I,
										uint64_cu *__hr_d_iN, 
										uint64_cu *__hr_d_iS, 
										uint64_cu *__hr_d_jE, 
										uint64_cu *__hr_d_jW,
										uint64_cu *__hr_d_dN, 
										uint64_cu *__hr_d_dS, 
										uint64_cu *__hr_d_dE, 
										uint64_cu *__hr_d_dW, 
										uint64_cu *__hr_d_c, 
										uint64_cu *__hr_d_I){
  /**** HIRACE STUFF ****/
  HiRaceDataWrap<fp> d_I         = _d_I;
  HiRaceDataWrap<fp> d_c         = _d_c    ;
  HiRaceDataWrap<int> d_iN        = _d_iN   ;
  HiRaceDataWrap<int> d_iS        = _d_iS   ;
  HiRaceDataWrap<int> d_jE        = _d_jE   ;
  HiRaceDataWrap<int> d_jW        = _d_jW   ;
  HiRaceDataWrap<fp> d_dN        = _d_dN   ;
  HiRaceDataWrap<fp> d_dS        = _d_dS   ;
  HiRaceDataWrap<fp> d_dW        = _d_dW   ;
  HiRaceDataWrap<fp> d_dE        = _d_dE   ;
   
  d_I.setMetadata(__hr_d_I);
  d_c.setMetadata(__hr_d_c);
  d_iN.setMetadata(__hr_d_iN);
  d_iS.setMetadata(__hr_d_iS);
  d_jE.setMetadata(__hr_d_jE);
  d_jW.setMetadata(__hr_d_jW);
  d_dN.setMetadata(__hr_d_dN);
  d_dS.setMetadata(__hr_d_dS);
  d_dW.setMetadata(__hr_d_dW);
  d_dE.setMetadata(__hr_d_dE);
   
  d_I.setScope(Scope::Global);
  d_c.setScope(Scope::Global);
  d_iN.setScope(Scope::Global);
  d_iS.setScope(Scope::Global);
  d_jE.setScope(Scope::Global);
  d_jW.setScope(Scope::Global);
  d_dN.setScope(Scope::Global);
  d_dS.setScope(Scope::Global);
  d_dW.setScope(Scope::Global);
  d_dE.setScope(Scope::Global);
  
  __syncthreads();
   
  #define d_I             d_I.registerCallsite(__LINE__,__FILE__)
  #define d_c             d_c.registerCallsite(__LINE__,__FILE__)
  #define d_iN            d_iN.registerCallsite(__LINE__,__FILE__)
  #define d_iS            d_iS.registerCallsite(__LINE__,__FILE__)
  #define d_jE            d_jE.registerCallsite(__LINE__,__FILE__)
  #define d_jW            d_jW.registerCallsite(__LINE__,__FILE__)
  #define d_dN            d_dN.registerCallsite(__LINE__,__FILE__)
  #define d_dS            d_dS.registerCallsite(__LINE__,__FILE__)
  #define d_dW            d_dW.registerCallsite(__LINE__,__FILE__)
  #define d_dE            d_dE.registerCallsite(__LINE__,__FILE__)
  /**** HIRACE STUFF ****/

	// indexes
    int bx = blockIdx.x;													// get current horizontal block index
	int tx = threadIdx.x;													// get current horizontal thread index
	int ei = bx*NUMBER_THREADS+tx;											// more threads than actual elements !!!
	int row;																// column, x position
	int col;																// row, y position

	// variables
	fp d_cN,d_cS,d_cW,d_cE;
	fp d_D;

	// figure out row/col location in new matrix
	row = (ei+1) % d_Nr - 1;												// (0-n) row
	col = (ei+1) / d_Nr + 1 - 1;											// (0-n) column
	if((ei+1) % d_Nr == 0){
		row = d_Nr - 1;
		col = col - 1;
	}

	if(ei<d_Ne){															// make sure that only threads matching jobs run

		// diffusion coefficent
		d_cN = d_c[ei];														// north diffusion coefficient
		d_cS = d_c[d_iS[row] + d_Nr*col];										// south diffusion coefficient
		d_cW = d_c[ei];														// west diffusion coefficient
		d_cE = d_c[row + d_Nr * d_jE[col]];									// east diffusion coefficient

		// divergence (equ 58)
		d_D = d_cN*d_dN[ei] + d_cS*d_dS[ei] + d_cW*d_dW[ei] + d_cE*d_dE[ei];// divergence

		// image update (equ 61) (every element of IMAGE)
		d_I[ei] = d_I[ei] + 0.25*d_lambda*d_D;								// updates image (based on input time step and divergence)

	}


  #undef d_I    
  #undef d_c    
  #undef d_iN   
  #undef d_iS   
  #undef d_jE   
  #undef d_jW   
  #undef d_dN   
  #undef d_dS   
  #undef d_dW   
  #undef d_dE   
}
