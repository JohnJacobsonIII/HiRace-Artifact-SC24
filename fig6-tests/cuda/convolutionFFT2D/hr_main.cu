#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample demonstrates how 2D convolutions
 * with very large kernel sizes
 * can be efficiently implemented
 * using FFT transformations.
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

// Helper functions for CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "hr_convolutionFFT2D_common.h"

//#include <HiRace.h>
#define HIRACE_SHADOW_DECL(NAME) hr_shadowt *__hr_metadata_ ## NAME ;
#define HIRACE_MALLOC(NAME, SIZE) hipMalloc((void **)&__hr_metadata_ ## NAME, SIZE * sizeof(hr_shadowt))
#define HIRACE_MEMSET(NAME, SIZE) hipMemset(__hr_metadata_ ## NAME, 0, SIZE * sizeof(hr_shadowt));
#define HIRACE_CUDA_FREE(NAME) hipFree(__hr_metadata_ ## NAME);
#define HIRACE_WRAP_DATA(TYPE, NAME) HiRaceDataWrap<TYPE> NAME(__hr_ ## NAME);
#define HIRACE_SET_DATA_GLOBAL(NAME) NAME.setMembers(__hr_ ## NAME, __hr_metadata_ ## NAME, Scope::Global, &bcount, &wcount, &swidx, 1, 0, 0);

using hr_shadowt = unsigned long long int;





////////////////////////////////////////////////////////////////////////////////
// Helper functions
////////////////////////////////////////////////////////////////////////////////
int snapTransformSize(int dataSize) {
  int hiBit;
  unsigned int lowPOT, hiPOT;

  dataSize = iAlignUp(dataSize, 16);

  for (hiBit = 31; hiBit >= 0; hiBit--)
    if (dataSize & (1U << hiBit)) {
      break;
    }

  lowPOT = 1U << hiBit;

  if (lowPOT == (unsigned int)dataSize) {
    return dataSize;
  }

  hiPOT = 1U << (hiBit + 1);

  if (hiPOT <= 1024) {
    return hiPOT;
  } else {
    return iAlignUp(dataSize, 512);
  }
}

float getRand(void) { return (float)(rand() % 16); }

bool test0(void) {
  float *h_Data, *h_Kernel, *h_ResultCPU, *h_ResultGPU;

  float *d_Data, *d_PaddedData, *d_Kernel, *d_PaddedKernel;

  fComplex *d_DataSpectrum, *d_KernelSpectrum;
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_SHADOW_DECL(d_Data)
  HIRACE_SHADOW_DECL(d_PaddedData)
  HIRACE_SHADOW_DECL(d_Kernel)
  HIRACE_SHADOW_DECL(d_PaddedKernel)
  HIRACE_SHADOW_DECL(d_DataSpectrum)
  HIRACE_SHADOW_DECL(d_KernelSpectrum)
  
  /************************/
  /***** HIRACE END *****/
  /************************/

  hipfftHandle fftPlanFwd, fftPlanInv;

  bool bRetVal;
  StopWatchInterface *hTimer = NULL;
  sdkCreateTimer(&hTimer);

  printf("Testing built-in R2C / C2R FFT-based convolution\n");
  const int kernelH = 7;
  const int kernelW = 6;
  const int kernelY = 3;
  const int kernelX = 4;
  const int dataH = 2000;
  const int dataW = 2000;
  const int fftH = snapTransformSize(dataH + kernelH - 1);
  const int fftW = snapTransformSize(dataW + kernelW - 1);

  printf("...allocating memory\n");
  h_Data = (float *)malloc(dataH * dataW * sizeof(float));
  h_Kernel = (float *)malloc(kernelH * kernelW * sizeof(float));
  h_ResultCPU = (float *)malloc(dataH * dataW * sizeof(float));
  h_ResultGPU = (float *)malloc(fftH * fftW * sizeof(float));

  checkCudaErrors(hipMalloc((void **)&d_Data, dataH * dataW * sizeof(float)));
  checkCudaErrors(
      hipMalloc((void **)&d_Kernel, kernelH * kernelW * sizeof(float)));

  checkCudaErrors(
      hipMalloc((void **)&d_PaddedData, fftH * fftW * sizeof(float)));
  checkCudaErrors(
      hipMalloc((void **)&d_PaddedKernel, fftH * fftW * sizeof(float)));

  checkCudaErrors(hipMalloc((void **)&d_DataSpectrum,
                             fftH * (fftW / 2 + 1) * sizeof(fComplex)));
  checkCudaErrors(hipMalloc((void **)&d_KernelSpectrum,
                             fftH * (fftW / 2 + 1) * sizeof(fComplex)));
  checkCudaErrors(hipMemset(d_KernelSpectrum, 0,
                             fftH * (fftW / 2 + 1) * sizeof(fComplex)));
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  checkCudaErrors(HIRACE_MALLOC(d_Data, dataH * dataW));
  checkCudaErrors(HIRACE_MALLOC(d_PaddedData, fftH * fftW));
  checkCudaErrors(HIRACE_MALLOC(d_Kernel, kernelH * kernelW));
  checkCudaErrors(HIRACE_MALLOC(d_PaddedKernel, fftH * fftW));
  checkCudaErrors(HIRACE_MALLOC(d_DataSpectrum, 2 * (fftH * (fftW / 2 + 1)))); // doubled for float2 handling
  checkCudaErrors(HIRACE_MALLOC(d_KernelSpectrum, 2 * (fftH * (fftW / 2 + 1))));
  
  HIRACE_MEMSET(d_Data, dataH * dataW)
  HIRACE_MEMSET(d_PaddedData, fftH * fftW)
  HIRACE_MEMSET(d_Kernel, kernelH * kernelW)
  HIRACE_MEMSET(d_PaddedKernel, fftH * fftW)
  HIRACE_MEMSET(d_DataSpectrum, 2 * (fftH * (fftW / 2 + 1)))
  HIRACE_MEMSET(d_KernelSpectrum, 2 * (fftH * (fftW / 2 + 1)))
  
  /************************/
  /***** HIRACE END *****/
  /************************/

  printf("...generating random input data\n");
  srand(2010);

  for (int i = 0; i < dataH * dataW; i++) {
    h_Data[i] = getRand();
  }

  for (int i = 0; i < kernelH * kernelW; i++) {
    h_Kernel[i] = getRand();
  }

  printf("...creating R2C & C2R FFT plans for %i x %i\n", fftH, fftW);
  checkCudaErrors(hipfftPlan2d(&fftPlanFwd, fftH, fftW, HIPFFT_R2C));
  checkCudaErrors(hipfftPlan2d(&fftPlanInv, fftH, fftW, HIPFFT_C2R));

  printf("...uploading to GPU and padding convolution kernel and input data\n");
  checkCudaErrors(hipMemcpy(d_Kernel, h_Kernel,
                             kernelH * kernelW * sizeof(float),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_Data, h_Data, dataH * dataW * sizeof(float),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(d_PaddedKernel, 0, fftH * fftW * sizeof(float)));
  checkCudaErrors(hipMemset(d_PaddedData, 0, fftH * fftW * sizeof(float)));

  padKernel(d_PaddedKernel, d_Kernel, fftH, fftW, kernelH, kernelW, kernelY,
            kernelX, __hr_metadata_d_PaddedKernel, __hr_metadata_d_Kernel);
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_MEMSET(d_Kernel, kernelH * kernelW)
  HIRACE_MEMSET(d_PaddedKernel, fftH * fftW)
  
  /************************/
  /***** HIRACE END *****/
  /************************/

  padDataClampToBorder(d_PaddedData, d_Data, fftH, fftW, dataH, dataW, kernelH,
                       kernelW, kernelY, kernelX, __hr_metadata_d_PaddedData, __hr_metadata_d_Data);
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_MEMSET(d_Data, dataH * dataW)
  HIRACE_MEMSET(d_PaddedData, fftH * fftW)
  
  /************************/
  /***** HIRACE END *****/
  /************************/

  // Not including kernel transformation into time measurement,
  // since convolution kernel is not changed very frequently
  printf("...transforming convolution kernel\n");
  checkCudaErrors(hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_PaddedKernel,
                               (hipfftComplex *)d_KernelSpectrum)); // HiRace: external library, just leak pointer here

  printf("...running GPU FFT convolution: ");
  checkCudaErrors(hipDeviceSynchronize());
  sdkResetTimer(&hTimer);
  sdkStartTimer(&hTimer);
  checkCudaErrors(hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_PaddedData,
                               (hipfftComplex *)d_DataSpectrum)); // HiRace: external library, just leak pointer here
  modulateAndNormalize(d_DataSpectrum, d_KernelSpectrum, fftH, fftW, 1, __hr_metadata_d_DataSpectrum, __hr_metadata_d_KernelSpectrum);
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_MEMSET(d_DataSpectrum, 2 * (fftH * (fftW / 2 + 1)))
  HIRACE_MEMSET(d_KernelSpectrum, 2 * (fftH * (fftW / 2 + 1)))
  
  /************************/
  /***** HIRACE END *****/
  /************************/

  checkCudaErrors(hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_DataSpectrum,
                               (hipfftReal *)d_PaddedData)); // HiRace: external library, just leak pointer here

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&hTimer);
  double gpuTime = sdkGetTimerValue(&hTimer);
  printf("%f MPix/s (%f ms)\n",
         (double)dataH * (double)dataW * 1e-6 / (gpuTime * 0.001), gpuTime);

  printf("...reading back GPU convolution results\n");
  checkCudaErrors(hipMemcpy(h_ResultGPU, d_PaddedData,
                             fftH * fftW * sizeof(float),
                             hipMemcpyDeviceToHost));

  printf("...running reference CPU convolution\n");
  convolutionClampToBorderCPU(h_ResultCPU, h_Data, h_Kernel, dataH, dataW,
                              kernelH, kernelW, kernelY, kernelX);

  printf("...comparing the results: ");
  double sum_delta2 = 0;
  double sum_ref2 = 0;
  double max_delta_ref = 0;

  for (int y = 0; y < dataH; y++)
    for (int x = 0; x < dataW; x++) {
      double rCPU = (double)h_ResultCPU[y * dataW + x];
      double rGPU = (double)h_ResultGPU[y * fftW + x];
      double delta = (rCPU - rGPU) * (rCPU - rGPU);
      double ref = rCPU * rCPU + rCPU * rCPU;

      if ((delta / ref) > max_delta_ref) {
        max_delta_ref = delta / ref;
      }

      sum_delta2 += delta;
      sum_ref2 += ref;
    }

  double L2norm = sqrt(sum_delta2 / sum_ref2);
  printf("rel L2 = %E (max delta = %E)\n", L2norm, sqrt(max_delta_ref));
  bRetVal = (L2norm < 1e-6) ? true : false;
  printf(bRetVal ? "L2norm Error OK\n" : "L2norm Error too high!\n");

  printf("...shutting down\n");
  sdkDeleteTimer(&hTimer);

  checkCudaErrors(hipfftDestroy(fftPlanInv));
  checkCudaErrors(hipfftDestroy(fftPlanFwd));
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_CUDA_FREE(d_Data)
  HIRACE_CUDA_FREE(d_PaddedData)
  HIRACE_CUDA_FREE(d_Kernel)
  HIRACE_CUDA_FREE(d_PaddedKernel)
  HIRACE_CUDA_FREE(d_DataSpectrum)
  HIRACE_CUDA_FREE(d_KernelSpectrum)
  
  /************************/
  /***** HIRACE END *****/
  /************************/

  checkCudaErrors(hipFree(d_DataSpectrum));
  checkCudaErrors(hipFree(d_KernelSpectrum));
  checkCudaErrors(hipFree(d_PaddedData));
  checkCudaErrors(hipFree(d_PaddedKernel));
  checkCudaErrors(hipFree(d_Data));
  checkCudaErrors(hipFree(d_Kernel));

  free(h_ResultGPU);
  free(h_ResultCPU);
  free(h_Data);
  free(h_Kernel);

  return bRetVal;
}

bool test1(void) {
  float *h_Data, *h_Kernel, *h_ResultCPU, *h_ResultGPU;

  float *d_Data, *d_Kernel, *d_PaddedData, *d_PaddedKernel;

  fComplex *d_DataSpectrum0, *d_KernelSpectrum0, *d_DataSpectrum,
      *d_KernelSpectrum;
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_SHADOW_DECL(d_Data)
  HIRACE_SHADOW_DECL(d_PaddedData)
  HIRACE_SHADOW_DECL(d_Kernel)
  HIRACE_SHADOW_DECL(d_PaddedKernel)
  HIRACE_SHADOW_DECL(d_DataSpectrum0)
  HIRACE_SHADOW_DECL(d_KernelSpectrum0)
  HIRACE_SHADOW_DECL(d_DataSpectrum)
  HIRACE_SHADOW_DECL(d_KernelSpectrum)
  
  /************************/
  /***** HIRACE END *****/
  /************************/


  hipfftHandle fftPlan;

  bool bRetVal;
  StopWatchInterface *hTimer = NULL;
  sdkCreateTimer(&hTimer);

  printf("Testing custom R2C / C2R FFT-based convolution\n");
  const uint fftPadding = 16;
  const int kernelH = 7;
  const int kernelW = 6;
  const int kernelY = 3;
  const int kernelX = 4;
  const int dataH = 2000;
  const int dataW = 2000;
  const int fftH = snapTransformSize(dataH + kernelH - 1);
  const int fftW = snapTransformSize(dataW + kernelW - 1);

  printf("...allocating memory\n");
  h_Data = (float *)malloc(dataH * dataW * sizeof(float));
  h_Kernel = (float *)malloc(kernelH * kernelW * sizeof(float));
  h_ResultCPU = (float *)malloc(dataH * dataW * sizeof(float));
  h_ResultGPU = (float *)malloc(fftH * fftW * sizeof(float));

  checkCudaErrors(hipMalloc((void **)&d_Data, dataH * dataW * sizeof(float)));
  checkCudaErrors(
      hipMalloc((void **)&d_Kernel, kernelH * kernelW * sizeof(float)));

  checkCudaErrors(
      hipMalloc((void **)&d_PaddedData, fftH * fftW * sizeof(float)));
  checkCudaErrors(
      hipMalloc((void **)&d_PaddedKernel, fftH * fftW * sizeof(float)));

  checkCudaErrors(hipMalloc((void **)&d_DataSpectrum0,
                             fftH * (fftW / 2) * sizeof(fComplex)));
  checkCudaErrors(hipMalloc((void **)&d_KernelSpectrum0,
                             fftH * (fftW / 2) * sizeof(fComplex)));
  checkCudaErrors(
      hipMalloc((void **)&d_DataSpectrum,
                 fftH * (fftW / 2 + fftPadding) * sizeof(fComplex)));
  checkCudaErrors(
      hipMalloc((void **)&d_KernelSpectrum,
                 fftH * (fftW / 2 + fftPadding) * sizeof(fComplex)));
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  checkCudaErrors(HIRACE_MALLOC(d_Data, dataH * dataW));
  checkCudaErrors(HIRACE_MALLOC(d_PaddedData, fftH * fftW));
  checkCudaErrors(HIRACE_MALLOC(d_Kernel, kernelH * kernelW));
  checkCudaErrors(HIRACE_MALLOC(d_PaddedKernel, fftH * fftW));
  checkCudaErrors(HIRACE_MALLOC(d_DataSpectrum0, 2 * (fftH * (fftW / 2)))); // doubled for float2 handling
  checkCudaErrors(HIRACE_MALLOC(d_KernelSpectrum0, 2 * (fftH * (fftW / 2))));
  checkCudaErrors(HIRACE_MALLOC(d_DataSpectrum, 2 * (fftH * (fftW / 2 + fftPadding))));
  checkCudaErrors(HIRACE_MALLOC(d_KernelSpectrum, 2 * (fftH * (fftW / 2 + fftPadding))));
  
  HIRACE_MEMSET(d_Data, dataH * dataW);
  HIRACE_MEMSET(d_PaddedData, fftH * fftW);
  HIRACE_MEMSET(d_Kernel, kernelH * kernelW);
  HIRACE_MEMSET(d_PaddedKernel, fftH * fftW);
  HIRACE_MEMSET(d_DataSpectrum0, 2 * (fftH * (fftW / 2)));
  HIRACE_MEMSET(d_KernelSpectrum0, 2 * (fftH * (fftW / 2)));
  HIRACE_MEMSET(d_DataSpectrum, 2 * (fftH * (fftW / 2 + fftPadding)));
  HIRACE_MEMSET(d_KernelSpectrum, 2 * (fftH * (fftW / 2 + fftPadding)));
  
  /************************/
  /***** HIRACE END *****/
  /************************/


  printf("...generating random input data\n");
  srand(2010);

  for (int i = 0; i < dataH * dataW; i++) {
    h_Data[i] = getRand();
  }

  for (int i = 0; i < kernelH * kernelW; i++) {
    h_Kernel[i] = getRand();
  }

  printf("...creating C2C FFT plan for %i x %i\n", fftH, fftW / 2);
  checkCudaErrors(hipfftPlan2d(&fftPlan, fftH, fftW / 2, HIPFFT_C2C));

  printf("...uploading to GPU and padding convolution kernel and input data\n");
  checkCudaErrors(hipMemcpy(d_Data, h_Data, dataH * dataW * sizeof(float),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_Kernel, h_Kernel,
                             kernelH * kernelW * sizeof(float),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(d_PaddedData, 0, fftH * fftW * sizeof(float)));
  checkCudaErrors(hipMemset(d_PaddedKernel, 0, fftH * fftW * sizeof(float)));

  padDataClampToBorder(d_PaddedData, d_Data, fftH, fftW, dataH, dataW, kernelH,
                       kernelW, kernelY, kernelX, __hr_metadata_d_PaddedData, __hr_metadata_d_Data);
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_MEMSET(d_Data, dataH * dataW);
  HIRACE_MEMSET(d_PaddedData, fftH * fftW);
  
  /************************/
  /***** HIRACE END *****/
  /************************/


  padKernel(d_PaddedKernel, d_Kernel, fftH, fftW, kernelH, kernelW, kernelY,
            kernelX, __hr_metadata_d_PaddedKernel, __hr_metadata_d_Kernel);
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_MEMSET(d_Kernel, kernelH * kernelW);
  HIRACE_MEMSET(d_PaddedKernel, fftH * fftW);
  
  /************************/
  /***** HIRACE END *****/
  /************************/


  // HIPFFT_BACKWARD works just as well...
  const int FFT_DIR = HIPFFT_FORWARD;

  // Not including kernel transformation into time measurement,
  // since convolution kernel is not changed very frequently
  printf("...transforming convolution kernel\n");
  checkCudaErrors(hipfftExecC2C(fftPlan, (hipfftComplex *)d_PaddedKernel,
                               (hipfftComplex *)d_KernelSpectrum0, FFT_DIR));
  spPostprocess2D(d_KernelSpectrum, d_KernelSpectrum0, fftH, fftW / 2,
                  fftPadding, FFT_DIR, __hr_metadata_d_KernelSpectrum, __hr_metadata_d_KernelSpectrum0);
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_MEMSET(d_KernelSpectrum0, 2 * (fftH * (fftW / 2)));
  HIRACE_MEMSET(d_KernelSpectrum, 2 * (fftH * (fftW / 2 + fftPadding)));
  
  /************************/
  /***** HIRACE END *****/
  /************************/


  printf("...running GPU FFT convolution: ");
  checkCudaErrors(hipDeviceSynchronize());
  sdkResetTimer(&hTimer);
  sdkStartTimer(&hTimer);

  checkCudaErrors(hipfftExecC2C(fftPlan, (hipfftComplex *)d_PaddedData,
                               (hipfftComplex *)d_DataSpectrum0, FFT_DIR));

  spPostprocess2D(d_DataSpectrum, d_DataSpectrum0, fftH, fftW / 2, fftPadding,
                  FFT_DIR, __hr_metadata_d_DataSpectrum, __hr_metadata_d_DataSpectrum0);
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_MEMSET(d_DataSpectrum0, 2 * (fftH * (fftW / 2)));
  HIRACE_MEMSET(d_DataSpectrum, 2 * (fftH * (fftW / 2 + fftPadding)));
  
  /************************/
  /***** HIRACE END *****/
  /************************/

  modulateAndNormalize(d_DataSpectrum, d_KernelSpectrum, fftH, fftW,
                       fftPadding, __hr_metadata_d_DataSpectrum, __hr_metadata_d_KernelSpectrum);
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_MEMSET(d_DataSpectrum, 2 * (fftH * (fftW / 2 + fftPadding)));
  HIRACE_MEMSET(d_KernelSpectrum, 2 * (fftH * (fftW / 2 + fftPadding)));
  
  /************************/
  /***** HIRACE END *****/
  /************************/

  spPreprocess2D(d_DataSpectrum0, d_DataSpectrum, fftH, fftW / 2, fftPadding,
                 -FFT_DIR, __hr_metadata_d_DataSpectrum0, __hr_metadata_d_DataSpectrum);
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_MEMSET(d_DataSpectrum0, 2 * (fftH * (fftW / 2)));
  HIRACE_MEMSET(d_DataSpectrum, 2 * (fftH * (fftW / 2 + fftPadding)));
  
  /************************/
  /***** HIRACE END *****/
  /************************/


  checkCudaErrors(hipfftExecC2C(fftPlan, (hipfftComplex *)d_DataSpectrum0,
                               (hipfftComplex *)d_PaddedData, -FFT_DIR));

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&hTimer);
  double gpuTime = sdkGetTimerValue(&hTimer);
  printf("%f MPix/s (%f ms)\n",
         (double)dataH * (double)dataW * 1e-6 / (gpuTime * 0.001), gpuTime);

  printf("...reading back GPU FFT results\n");
  checkCudaErrors(hipMemcpy(h_ResultGPU, d_PaddedData,
                             fftH * fftW * sizeof(float),
                             hipMemcpyDeviceToHost));

  printf("...running reference CPU convolution\n");
  convolutionClampToBorderCPU(h_ResultCPU, h_Data, h_Kernel, dataH, dataW,
                              kernelH, kernelW, kernelY, kernelX);

  printf("...comparing the results: ");
  double sum_delta2 = 0;
  double sum_ref2 = 0;
  double max_delta_ref = 0;

  for (int y = 0; y < dataH; y++)
    for (int x = 0; x < dataW; x++) {
      double rCPU = (double)h_ResultCPU[y * dataW + x];
      double rGPU = (double)h_ResultGPU[y * fftW + x];
      double delta = (rCPU - rGPU) * (rCPU - rGPU);
      double ref = rCPU * rCPU + rCPU * rCPU;

      if ((delta / ref) > max_delta_ref) {
        max_delta_ref = delta / ref;
      }

      sum_delta2 += delta;
      sum_ref2 += ref;
    }

  double L2norm = sqrt(sum_delta2 / sum_ref2);
  printf("rel L2 = %E (max delta = %E)\n", L2norm, sqrt(max_delta_ref));
  bRetVal = (L2norm < 1e-6) ? true : false;
  printf(bRetVal ? "L2norm Error OK\n" : "L2norm Error too high!\n");

  printf("...shutting down\n");
  sdkDeleteTimer(&hTimer);
  checkCudaErrors(hipfftDestroy(fftPlan));
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_CUDA_FREE(d_Data)
  HIRACE_CUDA_FREE(d_PaddedData)
  HIRACE_CUDA_FREE(d_Kernel)
  HIRACE_CUDA_FREE(d_PaddedKernel)
  HIRACE_CUDA_FREE(d_DataSpectrum0)
  HIRACE_CUDA_FREE(d_KernelSpectrum0)
  HIRACE_CUDA_FREE(d_DataSpectrum)
  HIRACE_CUDA_FREE(d_KernelSpectrum)
  
  /************************/
  /***** HIRACE END *****/
  /************************/


  checkCudaErrors(hipFree(d_KernelSpectrum));
  checkCudaErrors(hipFree(d_DataSpectrum));
  checkCudaErrors(hipFree(d_KernelSpectrum0));
  checkCudaErrors(hipFree(d_DataSpectrum0));
  checkCudaErrors(hipFree(d_PaddedKernel));
  checkCudaErrors(hipFree(d_PaddedData));
  checkCudaErrors(hipFree(d_Kernel));
  checkCudaErrors(hipFree(d_Data));

  free(h_ResultGPU);
  free(h_ResultCPU);
  free(h_Kernel);
  free(h_Data);

  return bRetVal;
}

bool test2(void) {
  float *h_Data, *h_Kernel, *h_ResultCPU, *h_ResultGPU;

  float *d_Data, *d_Kernel, *d_PaddedData, *d_PaddedKernel;

  fComplex *d_DataSpectrum0, *d_KernelSpectrum0;

  hipfftHandle fftPlan;
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_SHADOW_DECL(d_Data)
  HIRACE_SHADOW_DECL(d_PaddedData)
  HIRACE_SHADOW_DECL(d_Kernel)
  HIRACE_SHADOW_DECL(d_PaddedKernel)
  HIRACE_SHADOW_DECL(d_DataSpectrum0)
  HIRACE_SHADOW_DECL(d_KernelSpectrum0)
  
  /************************/
  /***** HIRACE END *****/
  /************************/


  bool bRetVal;
  StopWatchInterface *hTimer = NULL;
  sdkCreateTimer(&hTimer);

  printf("Testing updated custom R2C / C2R FFT-based convolution\n");
  const int kernelH = 7;
  const int kernelW = 6;
  const int kernelY = 3;
  const int kernelX = 4;
  const int dataH = 2000;
  const int dataW = 2000;
  const int fftH = snapTransformSize(dataH + kernelH - 1);
  const int fftW = snapTransformSize(dataW + kernelW - 1);

  printf("...allocating memory\n");
  h_Data = (float *)malloc(dataH * dataW * sizeof(float));
  h_Kernel = (float *)malloc(kernelH * kernelW * sizeof(float));
  h_ResultCPU = (float *)malloc(dataH * dataW * sizeof(float));
  h_ResultGPU = (float *)malloc(fftH * fftW * sizeof(float));

  checkCudaErrors(hipMalloc((void **)&d_Data, dataH * dataW * sizeof(float)));
  checkCudaErrors(
      hipMalloc((void **)&d_Kernel, kernelH * kernelW * sizeof(float)));

  checkCudaErrors(
      hipMalloc((void **)&d_PaddedData, fftH * fftW * sizeof(float)));
  checkCudaErrors(
      hipMalloc((void **)&d_PaddedKernel, fftH * fftW * sizeof(float)));

  checkCudaErrors(hipMalloc((void **)&d_DataSpectrum0,
                             fftH * (fftW / 2) * sizeof(fComplex)));
  checkCudaErrors(hipMalloc((void **)&d_KernelSpectrum0,
                             fftH * (fftW / 2) * sizeof(fComplex)));
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  checkCudaErrors(HIRACE_MALLOC(d_Data, dataH * dataW));
  checkCudaErrors(HIRACE_MALLOC(d_PaddedData, fftH * fftW));
  checkCudaErrors(HIRACE_MALLOC(d_Kernel, kernelH * kernelW));
  checkCudaErrors(HIRACE_MALLOC(d_PaddedKernel, fftH * fftW));
  checkCudaErrors(HIRACE_MALLOC(d_DataSpectrum0, 2 * (fftH * (fftW / 2)))); // doubled for float2 handling
  checkCudaErrors(HIRACE_MALLOC(d_KernelSpectrum0, 2 * (fftH * (fftW / 2))));
  
  HIRACE_MEMSET(d_Data, dataH * dataW)
  HIRACE_MEMSET(d_PaddedData, fftH * fftW)
  HIRACE_MEMSET(d_Kernel, kernelH * kernelW)
  HIRACE_MEMSET(d_PaddedKernel, fftH * fftW)
  HIRACE_MEMSET(d_DataSpectrum0, 2 * (fftH * (fftW / 2)))
  HIRACE_MEMSET(d_KernelSpectrum0, 2 * (fftH * (fftW / 2)))
  
  /************************/
  /***** HIRACE END *****/
  /************************/


  printf("...generating random input data\n");
  srand(2010);

  for (int i = 0; i < dataH * dataW; i++) {
    h_Data[i] = getRand();
  }

  for (int i = 0; i < kernelH * kernelW; i++) {
    h_Kernel[i] = getRand();
  }

  printf("...creating C2C FFT plan for %i x %i\n", fftH, fftW / 2);
  checkCudaErrors(hipfftPlan2d(&fftPlan, fftH, fftW / 2, HIPFFT_C2C));

  printf("...uploading to GPU and padding convolution kernel and input data\n");
  checkCudaErrors(hipMemcpy(d_Data, h_Data, dataH * dataW * sizeof(float),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_Kernel, h_Kernel,
                             kernelH * kernelW * sizeof(float),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(d_PaddedData, 0, fftH * fftW * sizeof(float)));
  checkCudaErrors(hipMemset(d_PaddedKernel, 0, fftH * fftW * sizeof(float)));

  padDataClampToBorder(d_PaddedData, d_Data, fftH, fftW, dataH, dataW, kernelH,
                       kernelW, kernelY, kernelX, __hr_metadata_d_PaddedData, __hr_metadata_d_Data);
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_MEMSET(d_Data, dataH * dataW)
  HIRACE_MEMSET(d_PaddedData, fftH * fftW)
  
  /************************/
  /***** HIRACE END *****/
  /************************/


  padKernel(d_PaddedKernel, d_Kernel, fftH, fftW, kernelH, kernelW, kernelY,
            kernelX, __hr_metadata_d_PaddedKernel, __hr_metadata_d_Kernel);
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_MEMSET(d_Kernel, kernelH * kernelW)
  HIRACE_MEMSET(d_PaddedKernel, fftH * fftW)
  
  /************************/
  /***** HIRACE END *****/
  /************************/


  // HIPFFT_BACKWARD works just as well...
  const int FFT_DIR = HIPFFT_FORWARD;

  // Not including kernel transformation into time measurement,
  // since convolution kernel is not changed very frequently
  printf("...transforming convolution kernel\n");
  checkCudaErrors(hipfftExecC2C(fftPlan, (hipfftComplex *)d_PaddedKernel,
                               (hipfftComplex *)d_KernelSpectrum0, FFT_DIR));

  printf("...running GPU FFT convolution: ");
  checkCudaErrors(hipDeviceSynchronize());
  sdkResetTimer(&hTimer);
  sdkStartTimer(&hTimer);

  checkCudaErrors(hipfftExecC2C(fftPlan, (hipfftComplex *)d_PaddedData,
                               (hipfftComplex *)d_DataSpectrum0, FFT_DIR));
  spProcess2D(d_DataSpectrum0, d_DataSpectrum0, d_KernelSpectrum0, fftH,
              fftW / 2, FFT_DIR, __hr_metadata_d_DataSpectrum0, __hr_metadata_d_DataSpectrum0, __hr_metadata_d_KernelSpectrum0);
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_MEMSET(d_DataSpectrum0, 2 * (fftH * (fftW / 2)))
  HIRACE_MEMSET(d_KernelSpectrum0, 2 * (fftH * (fftW / 2)))
  
  /************************/
  /***** HIRACE END *****/
  /************************/

  checkCudaErrors(hipfftExecC2C(fftPlan, (hipfftComplex *)d_DataSpectrum0,
                               (hipfftComplex *)d_PaddedData, -FFT_DIR));

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&hTimer);
  double gpuTime = sdkGetTimerValue(&hTimer);
  printf("%f MPix/s (%f ms)\n",
         (double)dataH * (double)dataW * 1e-6 / (gpuTime * 0.001), gpuTime);

  printf("...reading back GPU FFT results\n");
  checkCudaErrors(hipMemcpy(h_ResultGPU, d_PaddedData,
                             fftH * fftW * sizeof(float),
                             hipMemcpyDeviceToHost));

  printf("...running reference CPU convolution\n");
  convolutionClampToBorderCPU(h_ResultCPU, h_Data, h_Kernel, dataH, dataW,
                              kernelH, kernelW, kernelY, kernelX);

  printf("...comparing the results: ");
  double sum_delta2 = 0;
  double sum_ref2 = 0;
  double max_delta_ref = 0;

  for (int y = 0; y < dataH; y++) {
    for (int x = 0; x < dataW; x++) {
      double rCPU = (double)h_ResultCPU[y * dataW + x];
      double rGPU = (double)h_ResultGPU[y * fftW + x];
      double delta = (rCPU - rGPU) * (rCPU - rGPU);
      double ref = rCPU * rCPU + rCPU * rCPU;

      if ((delta / ref) > max_delta_ref) {
        max_delta_ref = delta / ref;
      }

      sum_delta2 += delta;
      sum_ref2 += ref;
    }
  }

  double L2norm = sqrt(sum_delta2 / sum_ref2);
  printf("rel L2 = %E (max delta = %E)\n", L2norm, sqrt(max_delta_ref));
  bRetVal = (L2norm < 1e-6) ? true : false;
  printf(bRetVal ? "L2norm Error OK\n" : "L2norm Error too high!\n");

  printf("...shutting down\n");
  sdkDeleteTimer(&hTimer);
  checkCudaErrors(hipfftDestroy(fftPlan));
  
  /************************/
  /***** HIRACE START *****/
  /************************/
  
  HIRACE_CUDA_FREE(d_Data)
  HIRACE_CUDA_FREE(d_PaddedData)
  HIRACE_CUDA_FREE(d_Kernel)
  HIRACE_CUDA_FREE(d_PaddedKernel)
  HIRACE_CUDA_FREE(d_DataSpectrum0)
  HIRACE_CUDA_FREE(d_KernelSpectrum0)
  
  /************************/
  /***** HIRACE END *****/
  /************************/


  checkCudaErrors(hipFree(d_KernelSpectrum0));
  checkCudaErrors(hipFree(d_DataSpectrum0));
  checkCudaErrors(hipFree(d_PaddedKernel));
  checkCudaErrors(hipFree(d_PaddedData));
  checkCudaErrors(hipFree(d_Kernel));
  checkCudaErrors(hipFree(d_Data));

  free(h_ResultGPU);
  free(h_ResultCPU);
  free(h_Kernel);
  free(h_Data);

  return bRetVal;
}

int main(int argc, char **argv) {
  printf("[%s] - Starting...\n", argv[0]);

  // Use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  findCudaDevice(argc, (const char **)argv);

  int nFailures = 0;

  if (!test0()) {
    nFailures++;
  }

  if (!test1()) {
    nFailures++;
  }

  if (!test2()) {
    nFailures++;
  }

  printf("Test Summary: %d errors\n", nFailures);

  if (nFailures > 0) {
    printf("Test failed!\n");
    exit(EXIT_FAILURE);
  }

  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}
