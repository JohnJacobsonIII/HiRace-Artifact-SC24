#include "hip/hip_runtime.h"
/* This file is part of the Indigo benchmark suite version 1.1.

Copyright 2022, Texas State University

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice,
   this list of conditions and the following disclaimer.
2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.
3. Neither the name of the copyright holder nor the names of its contributors
   may be used to endorse or promote products derived from this software
   without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.

Contributors: Yiqian Liu, Noushin Azami, Corbin Walters, and Martin Burtscher

URL: The latest version of the Indigo benchmark suite is available at
https://cs.txstate.edu/~burtscher/research/IndigoSuite/.
 */

typedef int data_t;
#include "indigo_cuda.h"

__global__ void test_kernel(int* nindex, int* nlist, data_t* __hr_data1, data_t* __hr_data2, int numv, hr_shadowt* __hr_metadata_data1, hr_shadowt* __hr_metadata_data2)
{
  /************************/
  /***** HIRACE START *****/
  /************************/
  unsigned __hr_bcount = 0, __hr_wcount = 0, __hr_swidx = 0;
  HIRACE_WRAP_DATA(data_t,data1)
  HIRACE_WRAP_DATA(data_t,data2)
  HIRACE_SET_DATA_GLOBAL(data1)
  HIRACE_SET_DATA_GLOBAL(data2)
  #define data1 data1.registerCallsite(__LINE__,__FILE__)
  #define data2 data2.registerCallsite(__LINE__,__FILE__)
  /************************/
  /***** HIRACE END *****/
  /************************/
  __shared__ data_t __hr_s_carry[32];
  /************************/
  /***** HIRACE START *****/
  /************************/
  int __hr_in_block_tid = threadIdx.x
  + threadIdx.y * blockDim.x
  + threadIdx.z * (blockDim.x * blockDim.y);
  int __hr_block_size = blockDim.x * blockDim.y * blockDim.z;
  int __hr_tid = __hr_in_block_tid
  + (blockIdx.x * __hr_block_size) // add a full block for each x step in grid
  + (blockIdx.y * gridDim.x * __hr_block_size) // a row of blocks to step in y
  + (blockIdx.z * gridDim.x * gridDim.y * __hr_block_size); // a square of blocks to step z
  int __hr_bid = blockIdx.x
  + blockIdx.y * gridDim.x
  + blockIdx.z * (gridDim.x * gridDim.y);
  HiRaceDataWrap<int> s_carry(__hr_s_carry);
  __shared__ hr_shadowt* __hr_metadata_s_carry;
  if (__hr_bid == 0) { // only check one block
  if (__hr_tid == 0) { // malloc with a single thread
  __hr_metadata_s_carry = new hr_shadowt[32];
  if(__hr_metadata_s_carry == NULL) { printf("HiRace: can't malloc shared metadata\n"); }
}
__syncthreads();
int __hr_size = 32;
// initialize the metadata to 0
unsigned __hr_offset = 0;
for (int i=0;i<__hr_size;i++) {
  unsigned __hr_idx = __hr_in_block_tid + __hr_offset;
  if(__hr_idx < __hr_size) __hr_metadata_s_carry[__hr_idx] = 0;
  __hr_offset += __hr_block_size;
}
__syncthreads();
s_carry.setMembers(__hr_s_carry,
__hr_metadata_s_carry,
Scope::Block,
&__hr_bcount,
&__hr_wcount,
&__hr_swidx,
1,0,0);
}

#define s_carry s_carry.registerCallsite(__LINE__,__FILE__)
/************************/
/***** HIRACE END *****/
/************************/
int lane = threadIdx.x % 32;
int warp = threadIdx.x / 32;
if (warp == 0) s_carry[lane] = 0;
__syncthreads();

for (int i = blockIdx.x; i < numv; i += gridDim.x) {
int beg = nindex[i];
int end = nindex[i + 1];
data_t val = 0;
for (int j = beg + threadIdx.x; j < end; j += blockDim.x) {
  int nei = nlist[j];
  val = max(val, data2[nei]);
}
val = max(val, __shfl_xor_sync(~0, val, 1));
val = max(val, __shfl_xor_sync(~0, val, 2));
val = max(val, __shfl_xor_sync(~0, val, 4));
val = max(val, __shfl_xor_sync(~0, val, 8));
val = max(val, __shfl_xor_sync(~0, val, 16));
if (lane == 0) s_carry[warp] = val;
__syncthreads();

if (warp == 0) {
  val = s_carry[lane];
  val = max(val, __shfl_xor_sync(~0, val, 1));
  val = max(val, __shfl_xor_sync(~0, val, 2));
  val = max(val, __shfl_xor_sync(~0, val, 4));
  val = max(val, __shfl_xor_sync(~0, val, 8));
  val = max(val, __shfl_xor_sync(~0, val, 16));
  if (lane == 0) {
    data1[0] = max(data1[0], val);
  }
}
}
/************************/
/***** HIRACE START *****/
/************************/
#undef data1
#undef data2
#undef s_carry
// delete global shadow of shared mem
__syncthreads();
if (__hr_tid == 0) { delete[] __hr_metadata_s_carry; }
/************************/
/***** HIRACE END *****/
/************************/
}

void serial_code(int* nindex, int* nlist, data_t* data1, data_t* data2, int numv)
{
for (int i = 0; i < numv; i++) {
int beg = nindex[i];
int end = nindex[i + 1];
for (int j = beg; j < end; j++) {
  int nei = nlist[j];
  data1[0] = max(data1[0], data2[nei]);
}
}
}

int verify_result(int* nindex, int* nlist, data_t* h_data1, data_t* h_data2, data_t* d_data1, data_t* d_data2, int numv, int nume, int blocks, int threadsperblock)
{
if (threadsperblock % 32 != 0) {
printf("Error: partial warps not supported\n");
return -1;
}
if (h_data1[0] == d_data1[0]) {
return 1;
} else {
return 0;
}
}
